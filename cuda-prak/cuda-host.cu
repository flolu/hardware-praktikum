#include "hip/hip_runtime.h"
#include "cuda-kernels.cu"

void copyImgCuda(unsigned char* img_in, unsigned char* img_out, int width, int height)
{
   unsigned char *img_in_dev, *img_out_dev;
   int size=width*height*4;
   // Speicher auf GPU allokieren
   hipMalloc((void**)&img_in_dev,size*sizeof(unsigned char));
   hipMalloc((void**)&img_out_dev,size*sizeof(unsigned char));
   // Konfiguration (Threads, Blöcke)
   dim3 threads(16,16);
   dim3 grid(width/threads.x+1,height/threads.y+1);
   // Kopieren der Daten auf Device
   hipMemcpy(img_in_dev,img_in,size*sizeof(unsigned char),hipMemcpyHostToDevice);
   // Ausführung copyKernel
   copyImgKernel<<<grid,threads>>>(img_in_dev,img_out_dev,width,height);
   // Kopieren der Daten auf Host
   hipMemcpy(img_out,img_out_dev,size*sizeof(unsigned char),hipMemcpyDeviceToHost);
   // Speicher auf GPU freigeben
   hipFree(img_in_dev);
   hipFree(img_out_dev);
}

void mirrorCuda(unsigned char* img_in, unsigned char* img_out, int width, int height)
{
   unsigned char *img_in_dev, *img_out_dev;
   int size=width*height*4;
   // Speicher auf GPU allokieren
   hipMalloc((void**)&img_in_dev,size*sizeof(unsigned char));
   hipMalloc((void**)&img_out_dev,size*sizeof(unsigned char));
   // Konfiguration (Threads, Blöcke)
   dim3 threads(16,16);
   dim3 grid(width/threads.x+1,height/threads.y+1);
   // Kopieren der Daten auf Device
   hipMemcpy(img_in_dev,img_in,size*sizeof(unsigned char),hipMemcpyHostToDevice);
   // Ausführung mirrorKernel
   mirrorKernel<<<grid,threads>>>(img_in_dev,img_out_dev,width,height);
   // Kopieren der Daten auf Host
   hipMemcpy(img_out,img_out_dev,size*sizeof(unsigned char),hipMemcpyDeviceToHost);
   // Speicher auf GPU freigeben
   hipFree(img_in_dev);
   hipFree(img_out_dev);
}


void linearTransformCuda(unsigned char* img_in, unsigned char* img_out, int width, int height,float alpha, float beta)
{
   unsigned char *img_in_dev, *img_out_dev;
   int size=width*height*4;
   // Speicher auf GPU allokieren
   hipMalloc((void**)&img_in_dev,size*sizeof(unsigned char));
   hipMalloc((void**)&img_out_dev,size*sizeof(unsigned char));
   // Konfiguration (Threads, Blöcke)
   dim3 threads(16,16);
   dim3 grid(width/threads.x+1,height/threads.y+1);
   // Kopieren der Daten auf Device
   hipMemcpy(img_in_dev,img_in,size*sizeof(unsigned char),hipMemcpyHostToDevice);
   // Ausführung linearKernel
   linearTransformKernel<<<grid,threads>>>(img_in_dev,img_out_dev,width,height,alpha,beta);
   // Kopieren der Daten auf Host
   hipMemcpy(img_out,img_out_dev,size*sizeof(unsigned char),hipMemcpyDeviceToHost);
   // Speicher auf GPU freigeben
   hipFree(img_in_dev);
   hipFree(img_out_dev);
}

void bwCuda(unsigned char* img_in, unsigned char* img_out, int width, int height)
{
   unsigned char *img_in_dev, *img_out_dev;
   int size=width*height*4;
   // Speicher auf GPU allokieren
   hipMalloc((void**)&img_in_dev,size*sizeof(unsigned char));
   hipMalloc((void**)&img_out_dev,size*sizeof(unsigned char));
   // Konfiguration (Threads, Blöcke)
   dim3 threads(16,16);
   dim3 grid(width/threads.x+1,height/threads.y+1);
   // Kopieren der Daten auf Device
   hipMemcpy(img_in_dev,img_in,size*sizeof(unsigned char),hipMemcpyHostToDevice);
   // Ausführung bwKernel
   bwKernel<<<grid,threads>>>(img_in_dev,img_out_dev,width,height);
   // Kopieren der Daten auf Host
   hipMemcpy(img_out,img_out_dev,size*sizeof(unsigned char),hipMemcpyDeviceToHost);
   // Speicher auf GPU freigeben
   hipFree(img_in_dev);
   hipFree(img_out_dev);
}

void sobelCuda(unsigned char* img_in, unsigned char* img_out, int width, int height)
{
  unsigned char *img_in_dev, *img_bw_dev, *img_out_dev;
  int size=width*height*4;
  // Speicher auf GPU allokieren (zusätzlich für bw-Bild)
  hipMalloc((void**)&img_in_dev,size*sizeof(unsigned char));
  hipMalloc((void**)&img_bw_dev,size*sizeof(unsigned char));
  hipMalloc((void**)&img_out_dev,size*sizeof(unsigned char));
  // Konfiguration (Threads, Blöcke)
  dim3 threads(16,16);
  dim3 grid(width/threads.x+1,height/threads.y+1);
  // Kopieren der Daten auf Device
  hipMemcpy(img_in_dev,img_in,size*sizeof(unsigned char),hipMemcpyHostToDevice);
  // Ausführung bwKernel, danach Ausführung sobelKernel
  bwKernel<<<grid,threads>>>(img_in_dev,img_bw_dev,width,height);
  sobelKernel<<<grid,threads>>>(img_bw_dev,img_out_dev,width,height);
  // Kopieren der Daten auf Host
  hipMemcpy(img_out,img_out_dev,size*sizeof(unsigned char),hipMemcpyDeviceToHost);
  // Speicher auf GPU freigeben (zusätzlich für bw-Bild)
  hipFree(img_in_dev);
  hipFree(img_bw_dev);
  hipFree(img_out_dev);
}