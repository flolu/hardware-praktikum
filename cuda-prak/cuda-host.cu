#include "hip/hip_runtime.h"
#include "cuda-kernels.cu"

void copyImgCuda(unsigned char* img_in, unsigned char* img_out, int width, int height)
{
   unsigned char *img_in_dev, *img_out_dev;
   int size=width*height*4;
   hipMalloc((void**)&img_in_dev,size*sizeof(unsigned char));
   hipMalloc((void**)&img_out_dev,size*sizeof(unsigned char));
   dim3 threads(16,16);
   dim3 grid(width/threads.x+1,height/threads.y+1);
   hipMemcpy(img_in_dev,img_in,size*sizeof(unsigned char),hipMemcpyHostToDevice);
   copyImgKernel<<<grid,threads>>>(img_in_dev,img_out_dev,width,height);
   hipMemcpy(img_out,img_out_dev,size*sizeof(unsigned char),hipMemcpyDeviceToHost);
   hipFree(img_in_dev);
   hipFree(img_out_dev);
}

void mirrorCuda(unsigned char* img_in, unsigned char* img_out, int width, int height)
{
   unsigned char *img_in_dev, *img_out_dev;
   int size=width*height*4;
   hipMalloc((void**)&img_in_dev,size*sizeof(unsigned char));
   hipMalloc((void**)&img_out_dev,size*sizeof(unsigned char));
   dim3 threads(16,16);
   dim3 grid(width/threads.x+1,height/threads.y+1);
   hipMemcpy(img_in_dev,img_in,size*sizeof(unsigned char),hipMemcpyHostToDevice);
   mirrorKernel<<<grid,threads>>>(img_in_dev,img_out_dev,width,height);
   hipMemcpy(img_out,img_out_dev,size*sizeof(unsigned char),hipMemcpyDeviceToHost);
   hipFree(img_in_dev);
   hipFree(img_out_dev);
}


void linearTransformCuda(unsigned char* img_in, unsigned char* img_out, int width, int height,float alpha, float beta)
{
   unsigned char *img_in_dev, *img_out_dev;
   int size=width*height*4;
   hipMalloc((void**)&img_in_dev,size*sizeof(unsigned char));
   hipMalloc((void**)&img_out_dev,size*sizeof(unsigned char));
   dim3 threads(16,16);
   dim3 grid(width/threads.x+1,height/threads.y+1);
   hipMemcpy(img_in_dev,img_in,size*sizeof(unsigned char),hipMemcpyHostToDevice);
   linearTransformKernel<<<grid,threads>>>(img_in_dev,img_out_dev,width,height,alpha,beta);
   hipMemcpy(img_out,img_out_dev,size*sizeof(unsigned char),hipMemcpyDeviceToHost);
   hipFree(img_in_dev);
   hipFree(img_out_dev);
}

void bwCuda(unsigned char* img_in, unsigned char* img_out, int width, int height)
{
   unsigned char *img_in_dev, *img_out_dev;
   int size=width*height*4;
   hipMalloc((void**)&img_in_dev,size*sizeof(unsigned char));
   hipMalloc((void**)&img_out_dev,size*sizeof(unsigned char));
   dim3 threads(16,16);
   dim3 grid(width/threads.x+1,height/threads.y+1);
   hipMemcpy(img_in_dev,img_in,size*sizeof(unsigned char),hipMemcpyHostToDevice);
   bwKernel<<<grid,threads>>>(img_in_dev,img_out_dev,width,height);
   hipMemcpy(img_out,img_out_dev,size*sizeof(unsigned char),hipMemcpyDeviceToHost);
   hipFree(img_in_dev);
   hipFree(img_out_dev);
}

void sobelCuda(unsigned char* img_in, unsigned char* img_out, int width, int height)
{
   //TODO: Aufgabe 2.5 Kantendetektion Hostcode
   //0. bwCuda() Code kopieren und um Folgendes erweitern:

   //1. temporäres GPU Array definieren und mittels hipMalloc anlegen (verwenden Sie als Namen z.B. img_bw_dev)

   //2. bwKernel ausführen und in img_bw_dev schreiben

   //3. sobelKernel ausführen und von img_bw_dev lesen, schreiben in img_out_dev

   //4. img_bw_dev Array wieder frei geben mit hipFree

}

