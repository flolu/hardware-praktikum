#include "hip/hip_runtime.h"
#include "cuda-kernels.cu"

void copyImgCuda(unsigned char* img_in, unsigned char* img_out, int width, int height)
{
   unsigned char *img_in_dev, *img_out_dev;
   int size=width*height*4;
   hipMalloc((void**)&img_in_dev,size*sizeof(unsigned char));
   hipMalloc((void**)&img_out_dev,size*sizeof(unsigned char));
   dim3 threads(16,16);
   dim3 grid(width/threads.x+1,height/threads.y+1);
   hipMemcpy(img_in_dev,img_in,size*sizeof(unsigned char),hipMemcpyHostToDevice);
   copyImgKernel<<<grid,threads>>>(img_in_dev,img_out_dev,width,height);
   hipMemcpy(img_out,img_out_dev,size*sizeof(unsigned char),hipMemcpyDeviceToHost);
   hipFree(img_in_dev);
   hipFree(img_out_dev);
}

void mirrorCuda(unsigned char* img_in, unsigned char* img_out, int width, int height)
{
   unsigned char *img_in_dev, *img_out_dev;
   int size=width*height*4;
   hipMalloc((void**)&img_in_dev,size*sizeof(unsigned char));
   hipMalloc((void**)&img_out_dev,size*sizeof(unsigned char));
   dim3 threads(16,16);
   dim3 grid(width/threads.x+1,height/threads.y+1);
   hipMemcpy(img_in_dev,img_in,size*sizeof(unsigned char),hipMemcpyHostToDevice);
   mirrorKernel<<<grid,threads>>>(img_in_dev,img_out_dev,width,height);
   hipMemcpy(img_out,img_out_dev,size*sizeof(unsigned char),hipMemcpyDeviceToHost);
   hipFree(img_in_dev);
   hipFree(img_out_dev);
}


void linearTransformCuda(unsigned char* img_in, unsigned char* img_out, int width, int height,float alpha, float beta)
{
   unsigned char *img_in_dev, *img_out_dev;
   int size=width*height*4;
   hipMalloc((void**)&img_in_dev,size*sizeof(unsigned char));
   hipMalloc((void**)&img_out_dev,size*sizeof(unsigned char));
   dim3 threads(16,16);
   dim3 grid(width/threads.x+1,height/threads.y+1);
   hipMemcpy(img_in_dev,img_in,size*sizeof(unsigned char),hipMemcpyHostToDevice);
   linearTransformKernel<<<grid,threads>>>(img_in_dev,img_out_dev,width,height,alpha,beta);
   hipMemcpy(img_out,img_out_dev,size*sizeof(unsigned char),hipMemcpyDeviceToHost);
   hipFree(img_in_dev);
   hipFree(img_out_dev);
}

void bwCuda(unsigned char* img_in, unsigned char* img_out, int width, int height)
{
   unsigned char *img_in_dev, *img_out_dev;
   int size=width*height*4;
   hipMalloc((void**)&img_in_dev,size*sizeof(unsigned char));
   hipMalloc((void**)&img_out_dev,size*sizeof(unsigned char));
   dim3 threads(16,16);
   dim3 grid(width/threads.x+1,height/threads.y+1);
   hipMemcpy(img_in_dev,img_in,size*sizeof(unsigned char),hipMemcpyHostToDevice);
   bwKernel<<<grid,threads>>>(img_in_dev,img_out_dev,width,height);
   hipMemcpy(img_out,img_out_dev,size*sizeof(unsigned char),hipMemcpyDeviceToHost);
   hipFree(img_in_dev);
   hipFree(img_out_dev);
}

void sobelCuda(unsigned char* img_in, unsigned char* img_out, int width, int height)
{
  unsigned char *img_in_dev, *img_bw_dev, *img_out_dev;
  int size=width*height*4;
  hipMalloc((void**)&img_in_dev,size*sizeof(unsigned char));
  hipMalloc((void**)&img_bw_dev,size*sizeof(unsigned char));
  hipMalloc((void**)&img_out_dev,size*sizeof(unsigned char));
  dim3 threads(16,16);
  dim3 grid(width/threads.x+1,height/threads.y+1);
  hipMemcpy(img_in_dev,img_in,size*sizeof(unsigned char),hipMemcpyHostToDevice);
  bwKernel<<<grid,threads>>>(img_in_dev,img_bw_dev,width,height);
  sobelKernel<<<grid,threads>>>(img_bw_dev,img_out_dev,width,height);
  hipMemcpy(img_out,img_out_dev,size*sizeof(unsigned char),hipMemcpyDeviceToHost);
  hipFree(img_in_dev);
  hipFree(img_bw_dev);
  hipFree(img_out_dev);
}

