
#include <hip/hip_runtime.h>
#define getR(img) (((unsigned int)img&0xFF000000)>>24)
#define getG(img) (((unsigned int)img&0x00FF0000)>>16)
#define getB(img) (((unsigned int)img&0x0000FF00)>>8)
#define getA(img) (((unsigned int)img&0x000000FF)>>0)
#define output(r,g,b,a) (((unsigned int)r<<24)+((unsigned int)g<<16)+((unsigned int)b<<8)+((unsigned int)a<<0))

__global__ void copyImgKernel(unsigned int* img_in, unsigned int* img_out, int width, int height)
{
   int i = threadIdx.x+blockIdx.x*blockDim.x;
   int j = threadIdx.y+blockIdx.y*blockDim.y;

   if (i<width && j<height)
   {
      int adrIn=i+j*width;
      int adrOut=adrIn;
      unsigned int color = img_in[adrIn];
      unsigned int r = getR(color);
      unsigned int g = getG(color);
      unsigned int b = getB(color);
      unsigned int a = getA(color);

      img_out[adrOut] = output(r,g,b,a);
   }
}

__device__ unsigned char checkOverflow(float value) {
  if (value > 255) return 255;
  return (unsigned char)value;
}

__global__ void linearTransformKernel(unsigned int* img_in, unsigned int* img_out, int width, int height, float alpha, float beta)
 {
   int i = threadIdx.x+blockIdx.x * blockDim.x;
   int j = threadIdx.y+blockIdx.y * blockDim.y;

   if (i<width && j<height)
   {
      int adrIn=i+j*width;
      int adrOut=adrIn;
      unsigned int color = img_in[adrIn];
      unsigned int r = checkOverflow(alpha * getR(color) + beta);
      unsigned int g = checkOverflow(alpha * getG(color) + beta);
      unsigned int b = checkOverflow(alpha * getB(color) + beta);
      unsigned int a = getA(color);

      img_out[adrOut] = output(r,g,b,a);
   }
 }

__global__ void mirrorKernel(unsigned int* img_in, unsigned int* img_out, int width, int height)
{
  int i = threadIdx.x+blockIdx.x*blockDim.x;
  int j = threadIdx.y+blockIdx.y*blockDim.y;

  if (i<width && j<height)
  {
    int adrIn=i+j*width;
    int adrOut=adrIn;

    unsigned int color;
    if (i >= width/2) {
      color = img_in[(width - i) + j * width];
    } else {
      color = img_in[adrIn];
    }

    unsigned int r = getR(color);
    unsigned int g = getG(color);
    unsigned int b = getB(color);
    unsigned int a = getA(color);

    img_out[adrOut] = output(r,g,b,a);
  }
}

__global__ void bwKernel(unsigned int* img_in, unsigned int* img_out, int width, int height)
{
  int i = threadIdx.x+blockIdx.x*blockDim.x;
  int j = threadIdx.y+blockIdx.y*blockDim.y;

  if (i<width && j<height)
  {
     int adrIn=i+j*width;
     int adrOut=adrIn;
     unsigned int color = img_in[adrIn];
     unsigned int a = getA(color);
     unsigned char grey = (getR(color) + getG(color) + getB(color)) / 3;
     img_out[adrOut] = output(grey, grey, grey, a);
  }
}

__global__ void sobelKernel(unsigned int* img_in, unsigned int* img_out, int width, int height)
{
  int i = threadIdx.x+blockIdx.x*blockDim.x;
  int j = threadIdx.y+blockIdx.y*blockDim.y;

  if (i<width && j<height)
  {
    int adrIn=i+j*width;
    int adrOut=adrIn;
    unsigned int color = img_in[adrIn];
    unsigned int a = getA(color);
    unsigned char color_byte = 0;

    if (i > 0 && i < width - 1 && j > 0 && j < height - 1) {
      const float SY[3][3]={{-1,-2,-1},{0,0,0},{1,2,1}};
      const float SX[3][3]={{-1,0,1},{-2,0,2},{-1,0,1}};

      float horizontal = 0;
      float vertical = 0;
      for (int k = -1; k <= 1; k++) {
        for (int l = -1; l <= 1; l++) {
          int adr = i+k + (j+l)*width;
          int grey = getR(img_in[adr]);
          horizontal += SY[1+k][1+l] * grey;
          vertical += SX[1+k][1+l] * grey;
        }
      }

      float color = sqrt(horizontal*horizontal + vertical*vertical);
      if (color > 255) color = 255;
      color_byte = (unsigned char)color;
    }

    img_out[adrOut] = output(color_byte, color_byte, color_byte, a);
  }
}
