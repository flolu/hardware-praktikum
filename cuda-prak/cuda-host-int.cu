//Host-Code für Aufgabe 3.2: Host-Code für die Verwendung von unsigned int
#include "hip/hip_runtime.h"
#include "cuda-kernels-int.cu"

//Auskommentierter Code ist eine Kopie von cuda-host.cu
//Überarbeiten Sie ihn für die Verwendung mit unsigned int!

void copyImgCuda(unsigned int* img_in, unsigned int* img_out, int width, int height)
{
   unsigned int *img_in_dev, *img_out_dev;
   int size=width*height;
   hipMalloc((void**)&img_in_dev,size*sizeof(unsigned int));
   hipMalloc((void**)&img_out_dev,size*sizeof(unsigned int));
   dim3 threads(16,16);
   dim3 grid(width/threads.x+1,height/threads.y+1);
   hipMemcpy(img_in_dev,img_in,size*sizeof(unsigned int),hipMemcpyHostToDevice);
   copyImgKernel<<<grid,threads>>>(img_in_dev,img_out_dev,width,height);
   hipMemcpy(img_out,img_out_dev,size*sizeof(unsigned int),hipMemcpyDeviceToHost);
   hipFree(img_in_dev);
   hipFree(img_out_dev);
}

void mirrorCuda(unsigned int* img_in, unsigned int* img_out, int width, int height)
{
/*    unsigned int *img_in_dev, *img_out_dev;
   int size=width*height;
   hipMalloc((void**)&img_in_dev,size*sizeof(unsigned int));
   hipMalloc((void**)&img_out_dev,size*sizeof(unsigned int));
   dim3 threads(16,16);
   dim3 grid(width/threads.x+1,height/threads.y+1);
   hipMemcpy(img_in_dev,img_in,size*sizeof(unsigned int),hipMemcpyHostToDevice);
   mirrorKernel<<<grid,threads>>>(img_in_dev,img_out_dev,width,height);
   hipMemcpy(img_out,img_out_dev,size*sizeof(unsigned int),hipMemcpyDeviceToHost);
   hipFree(img_in_dev);
   hipFree(img_out_dev); */
}


void linearTransformCuda(unsigned int* img_in, unsigned int* img_out, int width, int height,float alpha, float beta)
{
   unsigned int *img_in_dev, *img_out_dev;
   int size=width*height;
   hipMalloc((void**)&img_in_dev,size*sizeof(unsigned int));
   hipMalloc((void**)&img_out_dev,size*sizeof(unsigned int));
   dim3 threads(16,16);
   dim3 grid(width/threads.x+1,height/threads.y+1);
   hipMemcpy(img_in_dev,img_in,size*sizeof(unsigned int),hipMemcpyHostToDevice);
   linearTransformKernel<<<grid,threads>>>(img_in_dev,img_out_dev,width,height,alpha,beta);
   hipMemcpy(img_out,img_out_dev,size*sizeof(unsigned int),hipMemcpyDeviceToHost);
   hipFree(img_in_dev);
   hipFree(img_out_dev);
}

void bwCuda(unsigned int* img_in, unsigned int* img_out, int width, int height)
{
  /*  unsigned int *img_in_dev, *img_out_dev;
   int size=width*height*4;
   hipMalloc((void**)&img_in_dev,size*sizeof(unsigned int));
   hipMalloc((void**)&img_out_dev,size*sizeof(unsigned int));
   dim3 threads(16,16);
   dim3 grid(width/threads.x+1,height/threads.y+1);
   hipMemcpy(img_in_dev,img_in,size*sizeof(unsigned int),hipMemcpyHostToDevice);
   bwKernel<<<grid,threads>>>(img_in_dev,img_out_dev,width,height);
   hipMemcpy(img_out,img_out_dev,size*sizeof(unsigned int),hipMemcpyDeviceToHost);
   hipFree(img_in_dev);
   hipFree(img_out_dev); */
}

void sobelCuda(unsigned int* img_in, unsigned int* img_out, int width, int height)
{
/*    unsigned int *img_in_dev, *img_bw_dev, *img_out_dev;
   int size=width*height*4;
   hipMalloc((void**)&img_in_dev,size*sizeof(unsigned int));
   hipMalloc((void**)&img_bw_dev,size*sizeof(unsigned int));
   hipMalloc((void**)&img_out_dev,size*sizeof(unsigned int));
   dim3 threads(16,16);
   dim3 grid(width/threads.x+1,height/threads.y+1);
   hipMemcpy(img_in_dev,img_in,size*sizeof(unsigned int),hipMemcpyHostToDevice);
   bwKernel<<<grid,threads>>>(img_in_dev,img_bw_dev,width,height);
   sobelKernel<<<grid,threads>>>(img_bw_dev,img_out_dev,width,height);
   hipMemcpy(img_out,img_out_dev,size*sizeof(unsigned int),hipMemcpyDeviceToHost);
   hipFree(img_in_dev);
   hipFree(img_out_dev);
   hipFree(img_bw_dev); */
}
